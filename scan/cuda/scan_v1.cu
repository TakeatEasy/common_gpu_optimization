#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <stdio.h>
#include <bits/stdc++.h>



const int ARR_SIZE = 32 * 1024 * 1024;
const int BLOCK_SIZE = 1024;

__global__ void scan_and_write_part_sum_kernel(const int32_t* input, int32_t* part,
                                          int32_t* output, size_t n,
                                          size_t part_num) {
  __shared__ int32_t shm[BLOCK_SIZE];
  for (size_t part_i = blockIdx.x; part_i < part_num; part_i += gridDim.x) {
    // store this part input to shm
    size_t index = part_i * blockDim.x + threadIdx.x;
    shm[threadIdx.x] = index < n ? input[index] : 0;
    __syncthreads();
    if (threadIdx.x == 0) {  // naive implemention
        int32_t acc = 0;
        for (size_t i = 0; i < blockDim.x; ++i) {
        acc += shm[i];
        shm[i] = acc;
        }
    }
    __syncthreads();
    // write result
    if (index < n) {
      output[index] = shm[threadIdx.x];
    }
    if (threadIdx.x == blockDim.x - 1) {
      part[part_i] = shm[threadIdx.x];
    }
  }
}

__global__ void scan_part_sum_kernel(int32_t* part, size_t part_num) {
    // calcute the total sum of input
    int32_t acc = 0;
    for (size_t i = 0; i < part_num; i++) {
        acc += part[i];
        part[i] = acc;
    }
}

__global__ void add_base_sum_kernel(int32_t* part, int32_t* output, size_t n, size_t part_num) {
    // add part sum for each index
    for (size_t part_i = blockIdx.x; part_i < part_num; part_i += gridDim.x) {
        if (part_i == 0) {
            continue;
        }
        int32_t index = part_i * blockDim.x + threadIdx.x;
        if (index < n) {
            output[index] += part[part_i - 1];
        }
    }
}


void scan_the_fan(const int32_t* input, int32_t* output, size_t n) {
    // for each block
    size_t part_size = BLOCK_SIZE;
    size_t part_num = (n + part_size - 1) / part_size;
    size_t block_num = std::min<size_t>(part_num, 128);

    int32_t* part = new int[part_num];
    memset(part, 0, part_num);

    int32_t* d_input;
    int32_t* d_output;
    int32_t* d_part;

    
    hipMalloc((void**)&d_input, ARR_SIZE * sizeof(int32_t));
    hipMalloc((void**)&d_output, ARR_SIZE * sizeof(int32_t));
    hipMalloc((void**)&d_part, part_num * sizeof(int32_t));
    
    hipMemcpy(d_input, input, ARR_SIZE * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, ARR_SIZE * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_part, part, part_num * sizeof(int32_t), hipMemcpyHostToDevice);

    scan_and_write_part_sum_kernel<<<block_num, part_size>>>(d_input, d_part, d_output, ARR_SIZE, part_num);

    scan_part_sum_kernel<<<1, 1>>>(d_part, part_num);

    add_base_sum_kernel<<<block_num, part_size>>>(d_part, d_output, ARR_SIZE, part_num);

    hipMemcpy(output, d_output, ARR_SIZE * sizeof(int32_t), hipMemcpyDeviceToHost);
    hipMemcpy(part, d_part, part_num * sizeof(int32_t), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_part);
}

bool check_result(int* input, int* output, int size) {
    int acc = 0;
    for (int i = 0; i < size; i++) {
        acc += input[i];
        if (output[i] != acc) {
            printf("index %d output is %d should be %d\n", i, output[i], acc);
            return false;
        }
    }
    return true;
}

int main() {

    int* input = new int[ARR_SIZE];
    int* output = new int[ARR_SIZE];

    memset(output, 0, ARR_SIZE);

    for (int i = 0; i < ARR_SIZE; i++) {
        input[i] = i;
    }

    scan_the_fan(input, output, ARR_SIZE);

    for (int i = 0; i < 100; i++) {
        printf("output: %d\n", output[i]);
    }

    if (check_result(input, output, ARR_SIZE)) {
        printf("Right answer\n");
    } else {
        printf("Wrong answer\n");
    }

    delete[] input;
    delete[] output;
}